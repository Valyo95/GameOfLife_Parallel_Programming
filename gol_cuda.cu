#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <assert.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "./gol_lib/gol_array.h"
#include "./gol_lib/functions.h"

#define DEBUG 1
#define INFO 1
#define STATUS 1
#define TIME 1
#define REDUCE_RATE 1
#define PRINT_INITIAL 0
#define PRINT_STEPS 0
#define PRINT_FINAL 0
#define DEFAULT_N 420
#define DEFAULT_M 420
#define MAX_LOOPS 500
#define CUDA_THREADS 4
#define CUDA_BLOCKS 64

void print_1d_array(short int * array, int N, int M);

__global__ void parallel_populate(short int* array1, short int* array2, int N, int M, int *no_change)
{

  uint worldSize = N * M;

  //Κάθε GPU core παίρνει ένα κέλι με την σειρά
  for (uint cellId = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
      cellId < worldSize;
      cellId += blockDim.x * gridDim.x) {

    uint x = cellId % N;
    uint yAbs = cellId - x;
    uint xLeft = (x + N - 1) % N;
    uint xRight = (x + 1) % N;
    uint yAbsUp = (yAbs + worldSize - N) % worldSize;
    uint yAbsDown = (yAbs + N) % worldSize;
 
    uint aliveCells = array1[xLeft + yAbsUp] + array1[x + yAbsUp]
      + array1[xRight + yAbsUp] + array1[xLeft + yAbs] + array1[xRight + yAbs]
      + array1[xLeft + yAbsDown] + array1[x + yAbsDown] + array1[xRight + yAbsDown];

    array2[x + yAbs] =
      aliveCells == 3 || (aliveCells == 2 && array1[x + yAbs]) ? 1 : 0;	
  	
  	if(array1[x + yAbs] != array2[x + yAbs])
  		*no_change = 0;
  }
}

double timedifference_msec(struct timeval t0, struct timeval t1)
{
    return ((t1.tv_sec - t0.tv_sec) * 1000.0f + (t1.tv_usec - t0.tv_usec) / 1000.0f)/1000.0f;
}


int main(int argc, char* argv[])
{
	//the following can be also given by the user (to do)
	int N,M;
	int max_loops = -1;

	int cudaBlocks = -1;
	int cudaThreads = -1;

	struct timeval start;
	struct timeval finish;

	gol_array* ga1;
	gol_array* ga2;
	int i;

	/* SECTION A
		Blocks computation
		Matrix allocation, initialization
	*/

	char* filename = NULL;

	N = -1;
	M = -1;

  	//Read matrix size and game of life grid
  	//Or use default values and randomly generate a game if no arguments are given
	i = 0;
  	while (++i < argc)
  	{
  		if ( !strcmp(argv[i], "-f") )
  		{
  			filename = argv[i+1];
  			i++;
  		}
  		else if ( !strcmp(argv[i], "-l") )
  		{
  			N = atoi(argv[i+1]);
  			i++;
  		}
  		else if ( !strcmp(argv[i], "-c") )
  		{
  			M = atoi(argv[i+1]);
  			i++;
  		}
  		else if ( !strcmp(argv[i], "-m") )
  		{
  			max_loops = atoi(argv[i+1]);
  			i++;
  		}
  		else if ( !strcmp(argv[i], "-b") )
  		{
  			cudaBlocks = atoi(argv[i+1]);
  			i++;
  		}
		else if ( !strcmp(argv[i], "-t") )
  		{
  			cudaThreads = atoi(argv[i+1]);
  			i++;
  		}
  	}

	if (N == -1 || M == -1)
	{
		N = DEFAULT_N;
		M = DEFAULT_M;

		if (INFO)
		{
			printf("Running with default matrix size %dx%d\n", N, M);
			printf("Usage : './gol_cuda -f <filename> -l <N> -c <M> -n <max_loops>\n");
		}
	}
	else
	{
		if (N == 0 || M == 0)
		{
			printf("Invalid arguments given!");	
			printf("Usage : './gol_cuda -f <filename> -l <N> -c <M> -n <max_loops>\n");
			printf("Aborting...\n");
		}
	}

	if (max_loops == -1)
	{
		max_loops = MAX_LOOPS;
	}

	if (cudaThreads == -1)
	{
		cudaThreads = CUDA_THREADS;
		
	}
	
	if (cudaBlocks == -1)
	{
		cudaBlocks = CUDA_BLOCKS;	
	}

	if ( INFO )
	{
		printf("N = %d, M = %d\n", N, M);
		printf("Running with max loops:  %d\n", max_loops);
		printf("Running with blocks:     %d\n", cudaBlocks);
		printf("Running with threads:    %d\n", cudaThreads);
	}

	//allocate and init two NxM gol_arrays
	ga1 = gol_array_init(N, M);
	ga2 = gol_array_init(N, M);


	if (filename != NULL) 
	{
		gol_array_read_file(filename, ga1);
	}
	else 
	{//no input file given, generate a random game array
		if (INFO)
		{
			printf("No input file given as argument\n");
			printf("Generating a random game of life array to play\n");
		}
		gol_array_generate(ga1);
	}
	
/*	if (PRINT_INITIAL)
	{
		printf("Printing initial array:\n\n");
		print_array(ga1->array, N, M);
		putchar('\n');	
	}
*/	
	short int* arr1;
	short int* arr2;
	int *cudaNoChange;

	hipMalloc((void **) &cudaNoChange, sizeof(int *));
	hipMalloc((void **) &arr1, N*M*sizeof(short int *));
	hipMalloc((void **) &arr2, N*M*sizeof(short int *));
	
	short int* oneDarray1 = (short int *) malloc(N*M*sizeof(short int));
	short int* oneDarray2 = (short int *) malloc(N*M*sizeof(short int));
		

	hipMemcpy(arr1, ga1->flat_array, sizeof(short int) * M * N, hipMemcpyHostToDevice);

	int count;

	int *no_change = (int *) malloc(1*sizeof(int));

	if (STATUS)
		printf("Starting the Game of Life\n");

	gettimeofday(&start, 0);

	hipMemcpy(oneDarray1, arr1, sizeof(short int) * M * N, hipMemcpyDeviceToHost);
/*	print_1d_array(oneDarray1, N,M);
	printf("\n\n\n\n");*/


for(count = 0; count < max_loops; count++) 
	{
		*no_change = 1;
		hipMemcpy(cudaNoChange, no_change, sizeof(int), hipMemcpyHostToDevice);

		hipMemcpy(arr1, oneDarray1, sizeof(short int) * M * N, hipMemcpyHostToDevice);
		hipMemcpy(arr2, oneDarray2, sizeof(short int) * M * N, hipMemcpyHostToDevice);

		parallel_populate<<<cudaBlocks,cudaThreads>>>(arr1,  arr2, N,  M, cudaNoChange);

		hipMemcpy(oneDarray1, arr1, sizeof(short int) * M * N, hipMemcpyDeviceToHost);
		hipMemcpy(oneDarray2, arr2, sizeof(short int) * M * N, hipMemcpyDeviceToHost);
		
		hipMemcpy(no_change, cudaNoChange, sizeof(int), hipMemcpyDeviceToHost);

		if (PRINT_STEPS) {
			print_1d_array(oneDarray1	, N, M);
			putchar('\n');
		}

		if(count % REDUCE_RATE == 0)
		{
			if(*no_change == 1)
			{
				printf("Terminating because there was no change at loop number %d\n", count);
				break;
			}
		}

		//swap arrays (array2 becomes array1)
		short int* temp;
		temp = oneDarray1;
		oneDarray1 = oneDarray2;
		oneDarray2 = temp;
	}

	if (*no_change == 0 && STATUS)
	{
		printf("Max loop number (%d) was reached. Terminating Game of Life\n", max_loops);
	}


	gettimeofday(&finish, 0);
	
	double local_elapsed = timedifference_msec(start, finish);
	if (INFO)
		printf("Time elapsed: %.3f seconds\n", local_elapsed);

	if (PRINT_FINAL)
	{
		print_1d_array(oneDarray2, N,M);
	}

	//free arrays
	gol_array_free(&ga1);
	gol_array_free(&ga2);

	return 0;

}

gol_array* gol_array_init(int lines, int columns)
{
	//allocate one big flat array, so as to make sure that the memory is continuous
	//in our 2 dimensional array
	short int* flat_array = (short int*) calloc(lines*columns, sizeof(short int));
	assert(flat_array != NULL);

	short int** array = (short int**) malloc(lines*sizeof(short int*));
	assert(array != NULL);
	int i;

	//make a 2 dimension array by pointing to our flat 1 dimensional array
	
	for (i=0; i<lines; i++)
	{
		array[i] = &(flat_array[columns*i]);
	}

	//allocate gol_array struct
	gol_array* new_gol_array = (gol_array*) malloc(sizeof(gol_array));
	new_gol_array->flat_array = flat_array;
	new_gol_array->array = array;
	new_gol_array->lines = lines;
	new_gol_array->columns = columns;

	return new_gol_array;
}



void gol_array_free(gol_array** gol_ar)
{
	gol_array* gol_ar_ptr = *gol_ar;

	free(gol_ar_ptr->flat_array);
	free(gol_ar_ptr->array);
	free(*gol_ar);
	*gol_ar = NULL;
}



void gol_array_read_input(gol_array* gol_ar)
{
	short int** array = gol_ar->array;
	int N = gol_ar->lines;
	int M = gol_ar->columns;

	int row, col;
	printf("Give row and column of 'alive' cells\n");
	printf("Row [1, %d]\n", N);
	printf("Col [1, %d]\n", M);
	printf("To stop, just give a non-positive row or column\n");

	while (1)
	{
		printf("\nRow: ");
		scanf("%d", &row);
		if (row <= 0)
			break;

		printf("Column: ");
		scanf("%d", &col);
		if (col <= 0)
			break;

		if (row > N || col > M)
		{
			printf("Invalid row or column! Try again\n");
			printf("Row [1, %d]\n", N);
			printf("Col [1, %d]\n", M);
		}
		else
		{
			array[row-1][col-1] = 1;
		}
	}
}



void gol_array_read_file(char* filename, gol_array* gol_ar)
{
	short int** array = gol_ar->array;
	int N = gol_ar->lines;
	int M = gol_ar->columns;

	char line[100];
	char copy[100];
	int counter = 0;
	int successful = 0;

	FILE* file = fopen(filename, "r");

	if (file == NULL) 
	{
		printf("Error opening file\n");
		return;
	}

	while (1)
	{
		counter++;

		if (fgets(line, 100, file) == NULL)
			break;

		/*ignore blank lines or lines that start with '#'*/
		if (strlen(line) == 0 || line[0] == '#')
			continue;

		//the line should contain just 2 numbers
		//which are the coordinates (row column) of an alive organism/cell
		char* token;
		int row,col;

		strcpy(copy, line);//keep a copy of the actual line
		//strtok messes the string up.. 
		
		//get row (first token of line)
		token = strtok(line, " ");

		if (token == NULL)
		{
			printf("Skipping invalid line (%d): '%s'\n",counter, copy);
			continue;
		}

		row = atoi(token);

		//get column
		token = strtok(NULL, " ");

		if (token == NULL)
		{
			printf("Skipping invalid line (%d): '%s'\n",counter, copy);
			continue;
		}

		col = atoi(token);

		//ignore invalid lines (row or column out of bounds)
		if (row < 1 || row > N || col < 1 || col > M)
		{
			printf("Invalid row or column\n");
			printf("Skipping invalid line (%d): '%s'\n",counter, copy);
			continue;
		}

		successful++;
		array[row-1][col-1] = 1;
	}

	printf("\nSuccesfully read %d coordinates\n", successful);
	fclose(file);
}


void gol_array_generate(gol_array* gol_ar)
{
	char datestr[9];
	char timestr[7];

	get_date_time_str(datestr, timestr);
	char filename[36];
	sprintf(filename, "generated_tests/rga_%s_%s", datestr, timestr);
	
	FILE* file = fopen(filename, "w");
	if (file == NULL)
	{
		printf("gol_array_generate error opening file!\n");
		exit(-1);
	}

	short int** array = gol_ar->array;
	int lines = gol_ar->lines;
	int columns = gol_ar->columns;

	srand(time(NULL));
	int alive_count = rand() % (lines*columns + 1);
	int i;

	for (i=0; i<alive_count; i++)
	{
		int x,y;

		x = rand() % lines;
		y = rand() % columns;

		fprintf(file, "%d %d\n", x+1, y+1);
		array[x][y] = 1;
	}

	fclose(file);
}

void print_1d_array(short int * array, int N, int M)
{
	int i, j;

	for (i=0; i<N; i++)
	{
		putchar('|');

		for (j=0; j<M; j++)
		{
			if (array[i*M+j] == 1)
				putchar('o');
			else
				putchar(' ');

			putchar('|');
		}

		putchar('\n');
	}	
}

void print_array(short int** array, int N, int M)
{
	int i, j;

	for (i=0; i<N; i++)
	{
		putchar('|');

		for (j=0; j<M; j++)
		{
			if (array[i][j] == 1)
				putchar('o');
			else
				putchar(' ');

			putchar('|');
		}

		putchar('\n');
	}
}


int populate(short int** array1, short int** array2, int N, int M, int i, int j)
{
	int no_change;
	//get the number of neighbours
	int neighbours_num = num_of_neighbours(array1, N, M, i, j);

	if (array1[i][j] == 1)//if its alive
	{
		if (neighbours_num < 2 || neighbours_num > 3)//0,1 or 4 to 8 neighbours
		{//the organism dies
			array2[i][j] = 0;
			no_change = 0;
		}
		else//2 or 3 neigbours. So the organism survives (no change)
		{
			array2[i][j] = 1;
			no_change = 1;
		}
	}
	else//if its dead
	{
		if (neighbours_num == 3)//3 neighbours
		{//a new organism is born
			array2[i][j] = 1;
			no_change = 0;
		}
		else
		{//still no organism (no change)
			array2[i][j] = 0;
			no_change = 1;
		}
	}	

	return no_change;
}


int num_of_neighbours(short int** array, int N, int M, int row, int col)
{
	int up_row = (row-1+N) % N;
	int down_row = (row+1) % N;
	int right_col = (col+1) % M;
	int left_col = (col-1+M) % M;

	//neighbours
	int n1,n2,n3,n4,n5,n6,n7,n8;

	//up
	n1 = array[up_row][left_col];
	n2 = array[up_row][col];
	n3 = array[up_row][right_col];

	//down
	n4 = array[down_row][left_col];
	n5 = array[down_row][col];
	n6 = array[down_row][right_col];

	//left-right
	n7 = array[row][left_col];
	n8 = array[row][right_col];

	return n1 + n2 + n3 + n4 + n5 + n6 + n7 + n8;
}


void print_neighbour_nums(short int** array, int N, int M)
{
	int i,j;

	for (i=0; i<N; i++)
	{
		for (j=0; j<M; j++)
		{
			printf(" %d", num_of_neighbours(array, N, M, i, j));
		}
		putchar('\n');
	}
}


void get_date_time_str(char* datestr, char* timestr)
{
	time_t t = time(NULL);
	struct tm tm = *localtime(&t);
	char day[3];
	char month[3];
	char hour[3];
	char minute[3];
	char second[3];

	if (tm.tm_mon + 1 > 9)
		sprintf(month, "%d",tm.tm_mon + 1);
	else
		sprintf(month, "0%d",tm.tm_mon + 1);

	if (tm.tm_mday > 9)
		sprintf(day, "%d",tm.tm_mday);
	else
		sprintf(day, "0%d",tm.tm_mday);

	if (tm.tm_hour > 9)
		sprintf(hour, "%d",tm.tm_hour);
	else
		sprintf(hour, "0%d",tm.tm_hour);

	if (tm.tm_min > 9)
		sprintf(minute, "%d",tm.tm_min);
	else
		sprintf(minute, "0%d",tm.tm_min);

	if (tm.tm_sec > 9)
		sprintf(second, "%d",tm.tm_sec);
	else
		sprintf(second, "0%d",tm.tm_sec);

	sprintf(datestr, "%d%s%s", tm.tm_year + 1900, month, day);
	sprintf(timestr, "%s%s%s", hour, minute, second);
}
