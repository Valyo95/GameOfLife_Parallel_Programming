#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "./gol_lib/gol_array.h"
#include "./gol_lib/functions.h"

#define DEBUG 1
#define INFO 1
#define STATUS 1
#define TIME 1
#define PRINT_INITIAL 1
#define PRINT_STEPS 1
#define PRINT_FINAL 1
#define DEFAULT_N 420
#define DEFAULT_M 420
#define MAX_LOOPS 200
#define THREADS 1024

__global__ void parallel_populate(short int** array1, short int** array2, int N, int M, int rpb, int cpb, int bpr, int bpc, int *no_change)
{
	int i,j;
	int row_start, row_end, col_start, col_end;
  	
  	int index = threadIdx.x + blockIdx.x* blockDim.x;
  	//we use the process's coordinates instead of its rank
  	//because mpi might have reordered the processes for better performance (virtual topology)
  	row_start =  (index/bpr) * rpb ;
  	row_end = row_start + rpb - 1;
  	col_start = (index%bpc)* cpb;
  	col_end = col_start + cpb - 1;


  	for (i=row_start + 1; i<= row_end - 1; i++) {
		for (j= col_start + 1; j <= col_end - 1; j++) {
				//for each cell/organism

				//for each cell/organism
				//see if there is a change
				//populate functions applies the game's rules
				//and returns 0 if a change occurs
				//get the number of neighbours
			int neighbours_num;

			int up_row = (i-1+N) % N;
			int down_row = (i+1) % N;
			int right_col = (j+1) % M;
			int left_col = (j-1+M) % M;

			//neighbours
			int n1,n2,n3,n4,n5,n6,n7,n8;

			//up
			n1 = array1[up_row][left_col];
			n2 = array1[up_row][j];
			n3 = array1[up_row][right_col];

			//down
			n4 = array1[down_row][left_col];
			n5 = array1[down_row][j];
			n6 = array1[down_row][right_col];

			//left-right
			n7 = array1[i][left_col];
			n8 = array1[i][right_col];

			neighbours_num = n1 + n2 + n3 + n4 + n5 + n6 + n7 + n8;


			if (array1[i][j] == 1)//if its alive
			{
				if (neighbours_num < 2 || neighbours_num > 3)//0,1 or 4 to 8 neighbours
				{//the organism dies
					array2[i][j] = 0;
					*no_change = 0;
				}
				else//2 or 3 neigbours. So the organism survives (no change)
				{
					array2[i][j] = 1;
					*no_change = 1;
				}
			}
			else//if its dead
			{
				if (neighbours_num == 3)//3 neighbours
				{//a new organism is born
					array2[i][j] = 1;
					*no_change = 0;
				}
				else
				{//still no organism (no change)
					array2[i][j] = 0;
					*no_change = 1;
				}
			}	
		}
	}
}


int main(int argc, char* argv[])
{
	//the following can be also given by the user (to do)
	int N,M;
	int max_loops = -1;
	int threads = -1;

	gol_array* ga1;
	gol_array* ga2;
	int i, j;

	/* SECTION A
		Blocks computation
		Matrix allocation, initialization
	*/

	char* filename = NULL;

	N = -1;
	M = -1;

  	//Read matrix size and game of life grid
  	//Or use default values and randomly generate a game if no arguments are given
	i = 0;
  	while (++i < argc)
  	{
  		if ( !strcmp(argv[i], "-f") )
  		{
  			filename = argv[i+1];
  			i++;
  		}
  		else if ( !strcmp(argv[i], "-l") )
  		{
  			N = atoi(argv[i+1]);
  			i++;
  		}
  		else if ( !strcmp(argv[i], "-c") )
  		{
  			M = atoi(argv[i+1]);
  			i++;
  		}
  		else if ( !strcmp(argv[i], "-m") )
  		{
  			max_loops = atoi(argv[i+1]);
  			i++;
  		}
		else if ( !strcmp(argv[i], "-t") )
  		{
  			threads = atoi(argv[i+1]);
  			i++;
  		}
  	}

	if (N == -1 || M == -1)
	{
		N = DEFAULT_N;
		M = DEFAULT_M;

		if (INFO)
		{
			printf("Running with default matrix size %dx%d\n", N, M);
			printf("Usage : './gol_cuda -f <filename> -l <N> -c <M> -n <max_loops>\n");
		}
	}
	else
	{
		if (N == 0 || M == 0)
		{
			printf("Invalid arguments given!");	
			printf("Usage : './gol_cuda -f <filename> -l <N> -c <M> -n <max_loops>\n");
			printf("Aborting...\n");
		}
	}

	if (max_loops == -1)
	{
		max_loops = MAX_LOOPS;
		
		if ( INFO )
			printf("Running with default max loops %d\n", max_loops);
	}

	if (threads == -1)
	{
		threads = THREADS;
		
		if ( INFO )
			printf("Running with default threads num %d\n", threads);
	}



	if (INFO)
		printf("N = %d\nM = %d\n", N, M);

	//Calculate block properties
	int line_div, col_div;
	double threads_sqrt = sqrt(threads);
	double div;
	int last_div_ok;

	if (threads_sqrt == floor(threads))
	{
		line_div = threads_sqrt;
		col_div = threads_sqrt;
	}
	else
	{
		last_div_ok = 1;
		i = 2;
		while ( i < (threads / 2) )
		{
			div = (float) threads / (float) i;

			if (floor(div) == div)
			{
				if ( ( abs((int) div - (int) (threads / div)) >= abs((int) last_div_ok - (int) (threads / last_div_ok)) )
					|| div == last_div_ok ) {
					break;
				}

				last_div_ok = (int) div;
			}

			i++;
		}

		if (last_div_ok == 1 && threads != 2 && threads != 1)
		{
			printf("Warning, threads num is a prime number and can't be devided well\n");
		}

		line_div = last_div_ok;
		col_div = threads / line_div;
	}

	int rows_per_block = N / line_div;
	int cols_per_block = M / col_div;
	int blocks_per_row = N / rows_per_block;
	int blocks_per_col = M / cols_per_block;

	if (INFO)
	{
		printf("line div : %d\n", line_div);
		printf("col div : %d\n", col_div);
		printf("rows_per_block: %d\n", rows_per_block);
		printf("cols_per_block: %d\n", cols_per_block);
	}

	//allocate and init two NxM gol_arrays
	ga1 = gol_array_init(N, M);
	ga2 = gol_array_init(N, M);


	if (filename != NULL) 
	{
		gol_array_read_file(filename, ga1);
	}
	else 
	{//no input file given, generate a random game array
		if (INFO)
		{
			printf("No input file given as argument\n");
			printf("Generating a random game of life array to play\n");
		}
		gol_array_generate(ga1);
	}
	
	if (PRINT_INITIAL)
	{
		printf("Printing initial array:\n\n");
		print_array(ga1->array, N, M);
		putchar('\n');	
	}




  	//Calculate this threads's boundaries
	int row_start, row_end, col_start, col_end;
  	
  	
/*
  	row_start = 
  	row_end = 
  	col_start = 
  	col_end = 
*/


	short int** array1 = ga1->array;
	short int** array2 = ga2->array;
	
	short int* arr1;
	short int* arr2;

	hipMalloc((void **) &arr1, N*M*sizeof(short int *));
	hipMalloc((void **) &arr2, N*M*sizeof(short int *));
	short int** arr33 = (short int **) malloc(N*sizeof(short int*));
	short int* arrAll = (short int *) malloc(N*M*sizeof(short int));

	for (int i = 0; i < N; ++i)
	{
		memcpy( arrAll + i*M*sizeof(short int) , array1[i], M*sizeof(short int));
	}

	hipMemcpy(arr1, arrAll, sizeof(short int) * M * N, hipMemcpyHostToDevice);

	for (int i = 0; i < N; ++i)
	{
		printf("i=%d\n", i);
		arr33[i] = (short int *) malloc(M*sizeof(short int));
		hipMemcpy(arr33 + i*M*sizeof(short int) , arr1 + i*M*sizeof(short int), sizeof(short int) * M, hipMemcpyDeviceToHost);
	}
	printf("hahahaha\n");
	print_array(arr33, N, M);
	return;

	int count;
	int no_change;
	int no_change_sum;

	double start, finish;

	if (STATUS)
		printf("Starting the Game of Life\n");

	start = time(NULL);

	for(count = 0; count < max_loops; count++) 
	{
		no_change = 1;


		if (PRINT_STEPS) {
			print_array(array2, N, M);
			putchar('\n');
		}

		//swap arrays (array2 becomes array1)
		short int** temp;
		temp = array1;
		array1 = array2;
		array2 = temp;

	}

	if (no_change == 0 && STATUS)
	{
		printf("Max loop number (%d) was reached. Terminating Game of Life\n", max_loops);
	}


	finish = time(NULL);

	double local_elapsed = finish - start;

	if (INFO)
		printf("Time elapsed: %f seconds\n", local_elapsed);

	if (PRINT_FINAL)
	{
		print_array(array1, N, M);
	}

	//free arrays
	gol_array_free(&ga1);
	gol_array_free(&ga2);

	return 0;

}

gol_array* gol_array_init(int lines, int columns)
{
	//allocate one big flat array, so as to make sure that the memory is continuous
	//in our 2 dimensional array
	short int* flat_array = (short int*) calloc(lines*columns, sizeof(short int));
	assert(flat_array != NULL);

	short int** array = (short int**) malloc(lines*sizeof(short int*));
	assert(array != NULL);
	int i;

	//make a 2 dimension array by pointing to our flat 1 dimensional array
	
	for (i=0; i<lines; i++)
	{
		array[i] = &(flat_array[columns*i]);
	}

	//allocate gol_array struct
	gol_array* new_gol_array = (gol_array*) malloc(sizeof(gol_array));
	new_gol_array->flat_array = flat_array;
	new_gol_array->array = array;
	new_gol_array->lines = lines;
	new_gol_array->columns = columns;

	return new_gol_array;
}



void gol_array_free(gol_array** gol_ar)
{
	gol_array* gol_ar_ptr = *gol_ar;

	free(gol_ar_ptr->flat_array);
	free(gol_ar_ptr->array);
	free(*gol_ar);
	*gol_ar = NULL;
}



void gol_array_read_input(gol_array* gol_ar)
{
	short int** array = gol_ar->array;
	int N = gol_ar->lines;
	int M = gol_ar->columns;

	int row, col;
	printf("Give row and column of 'alive' cells\n");
	printf("Row [1, %d]\n", N);
	printf("Col [1, %d]\n", M);
	printf("To stop, just give a non-positive row or column\n");

	while (1)
	{
		printf("\nRow: ");
		scanf("%d", &row);
		if (row <= 0)
			break;

		printf("Column: ");
		scanf("%d", &col);
		if (col <= 0)
			break;

		if (row > N || col > M)
		{
			printf("Invalid row or column! Try again\n");
			printf("Row [1, %d]\n", N);
			printf("Col [1, %d]\n", M);
		}
		else
		{
			array[row-1][col-1] = 1;
		}
	}
}



void gol_array_read_file(char* filename, gol_array* gol_ar)
{
	short int** array = gol_ar->array;
	int N = gol_ar->lines;
	int M = gol_ar->columns;

	char line[100];
	char copy[100];
	int counter = 0;
	int successful = 0;

	FILE* file = fopen(filename, "r");

	if (file == NULL) 
	{
		printf("Error opening file\n");
		return;
	}

	while (1)
	{
		counter++;

		if (fgets(line, 100, file) == NULL)
			break;

		/*ignore blank lines or lines that start with '#'*/
		if (strlen(line) == 0 || line[0] == '#')
			continue;

		//the line should contain just 2 numbers
		//which are the coordinates (row column) of an alive organism/cell
		char* token;
		int row,col;

		strcpy(copy, line);//keep a copy of the actual line
		//strtok messes the string up.. 
		
		//get row (first token of line)
		token = strtok(line, " ");

		if (token == NULL)
		{
			printf("Skipping invalid line (%d): '%s'\n",counter, copy);
			continue;
		}

		row = atoi(token);

		//get column
		token = strtok(NULL, " ");

		if (token == NULL)
		{
			printf("Skipping invalid line (%d): '%s'\n",counter, copy);
			continue;
		}

		col = atoi(token);

		//ignore invalid lines (row or column out of bounds)
		if (row < 1 || row > N || col < 1 || col > M)
		{
			printf("Invalid row or column\n");
			printf("Skipping invalid line (%d): '%s'\n",counter, copy);
			continue;
		}

		successful++;
		array[row-1][col-1] = 1;
	}

	printf("\nSuccesfully read %d coordinates\n", successful);
	fclose(file);
}


void gol_array_generate(gol_array* gol_ar)
{
	char datestr[9];
	char timestr[7];

	get_date_time_str(datestr, timestr);
	char filename[36];
	sprintf(filename, "generated_tests/rga_%s_%s", datestr, timestr);
	
	FILE* file = fopen(filename, "w");
	if (file == NULL)
	{
		printf("gol_array_generate error opening file!\n");
		exit(-1);
	}

	short int** array = gol_ar->array;
	int lines = gol_ar->lines;
	int columns = gol_ar->columns;

	srand(time(NULL));
	int alive_count = rand() % (lines*columns + 1);
	int i;

	for (i=0; i<alive_count; i++)
	{
		int x,y;

		x = rand() % lines;
		y = rand() % columns;

		fprintf(file, "%d %d\n", x+1, y+1);
		array[x][y] = 1;
	}

	fclose(file);
}

void print_array(short int** array, int N, int M)
{
	int i, j;

	for (i=0; i<N; i++)
	{
		putchar('|');

		for (j=0; j<M; j++)
		{
			if (array[i][j] == 1)
				putchar('o');
			else
				putchar(' ');

			putchar('|');
		}

		putchar('\n');
	}
}


int populate(short int** array1, short int** array2, int N, int M, int i, int j)
{
	int no_change;
	//get the number of neighbours
	int neighbours_num = num_of_neighbours(array1, N, M, i, j);

	if (array1[i][j] == 1)//if its alive
	{
		if (neighbours_num < 2 || neighbours_num > 3)//0,1 or 4 to 8 neighbours
		{//the organism dies
			array2[i][j] = 0;
			no_change = 0;
		}
		else//2 or 3 neigbours. So the organism survives (no change)
		{
			array2[i][j] = 1;
			no_change = 1;
		}
	}
	else//if its dead
	{
		if (neighbours_num == 3)//3 neighbours
		{//a new organism is born
			array2[i][j] = 1;
			no_change = 0;
		}
		else
		{//still no organism (no change)
			array2[i][j] = 0;
			no_change = 1;
		}
	}	

	return no_change;
}


int num_of_neighbours(short int** array, int N, int M, int row, int col)
{
	int up_row = (row-1+N) % N;
	int down_row = (row+1) % N;
	int right_col = (col+1) % M;
	int left_col = (col-1+M) % M;

	//neighbours
	int n1,n2,n3,n4,n5,n6,n7,n8;

	//up
	n1 = array[up_row][left_col];
	n2 = array[up_row][col];
	n3 = array[up_row][right_col];

	//down
	n4 = array[down_row][left_col];
	n5 = array[down_row][col];
	n6 = array[down_row][right_col];

	//left-right
	n7 = array[row][left_col];
	n8 = array[row][right_col];

	return n1 + n2 + n3 + n4 + n5 + n6 + n7 + n8;
}


void print_neighbour_nums(short int** array, int N, int M)
{
	int i,j;

	for (i=0; i<N; i++)
	{
		for (j=0; j<M; j++)
		{
			printf(" %d", num_of_neighbours(array, N, M, i, j));
		}
		putchar('\n');
	}
}


void get_date_time_str(char* datestr, char* timestr)
{
	time_t t = time(NULL);
	struct tm tm = *localtime(&t);
	char day[3];
	char month[3];
	char hour[3];
	char minute[3];
	char second[3];

	if (tm.tm_mon + 1 > 9)
		sprintf(month, "%d",tm.tm_mon + 1);
	else
		sprintf(month, "0%d",tm.tm_mon + 1);

	if (tm.tm_mday > 9)
		sprintf(day, "%d",tm.tm_mday);
	else
		sprintf(day, "0%d",tm.tm_mday);

	if (tm.tm_hour > 9)
		sprintf(hour, "%d",tm.tm_hour);
	else
		sprintf(hour, "0%d",tm.tm_hour);

	if (tm.tm_min > 9)
		sprintf(minute, "%d",tm.tm_min);
	else
		sprintf(minute, "0%d",tm.tm_min);

	if (tm.tm_sec > 9)
		sprintf(second, "%d",tm.tm_sec);
	else
		sprintf(second, "0%d",tm.tm_sec);

	sprintf(datestr, "%d%s%s", tm.tm_year + 1900, month, day);
	sprintf(timestr, "%s%s%s", hour, minute, second);
}